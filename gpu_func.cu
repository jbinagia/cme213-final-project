#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"

__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_result));
    return result;
}

__global__
void gpuGEMM(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double alpha, double beta,
           int M, int N, int K) {

    uint i = (blockIdx.y * blockDim.y) + threadIdx.y; // let this correspond to row index
    uint j = (blockIdx.x * blockDim.x) + threadIdx.x; // let this correspond to column index 

    if (i < M && j < N){ 
        C[j*M + i] = beta*C[j*M + i]; 
        for (int k = 0; k < K; k++){
            C[j*M + i] += alpha*A[k*M + i]*B[j*K + k]; 
        }
    }
}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double* alpha, double* beta,
           int M, int N, int K) {
    /* TODO: Write an efficient GEMM implementation on GPU */

    // here is where I need to implement the CUDA GEMM kernel
    // - need to dereference alpha and beta via *alpha before using them 
    // - A, B, C are all device arrays at this point 

    dim3 threadsPerBlock(8, 32);  // 256 threads
    int num_blocks_x = (N + threadsPerBlock.x - 1)/threadsPerBlock.x; // N is number of columns
    int num_blocks_y = (M + threadsPerBlock.y - 1)/threadsPerBlock.y; // M is number of rows
    dim3 numBlocks(num_blocks_x, num_blocks_y); 
    gpuGEMM<<<numBlocks, threadsPerBlock>>>(A, B, C, *alpha, *beta, M, N, K); 

    return 1;
}

// wrapper to automate allocating and transferring memory 
int wrapperGEMM(double* __restrict__ A, double* __restrict__ B,
           double* __restrict__ C, double* alpha, double* beta,
           int M, int N, int K) {

    double* dA;
    double* dB;
    double* dC;

    hipMalloc((void**)&dA, sizeof(double) * M * K);
    hipMalloc((void**)&dB, sizeof(double) * K * N);
    hipMalloc((void**)&dC, sizeof(double) * M * N);

    hipMemcpy(dA, A, sizeof(double) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(double) * K * N, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(double) * M * N, hipMemcpyHostToDevice);

    int err = myGEMM(dA, dB, dC, alpha, beta, M, N, K); 
    hipMemcpy(C, dC, sizeof(double) * M * N, hipMemcpyDeviceToHost);

    return 1;
}

